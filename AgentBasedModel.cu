#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sstream>
#include <iostream>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include "AgentLib.h"
#include <thrust/fill.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

using namespace std;

int main(int argc, char* argv[]){
    double seed;
    int numInfected = 0;
    int timeStep = 0;
    int popWidth = sqrt(population);
    int numLinksHetero = 0;
    int numLinksMSM = 0;
    int numLinksIDU = 0;
    int numUsers = 0;
    int* users;
    int* coordD;
    int* dummyInts;
    agent* agentList;
    heteroNetRanges* hnrH = {&hnr};
    MSMNetRanges* msmrH = {&msmr};
    IDUNetRanges* idurH = {&idur};
    heteroNetRanges* hnrD;
    MSMNetRanges* msmrD;
    IDUNetRanges* idurD;
    int2 *hNetEdges;
    int2 *MSMNetEdges;
    int2 *IDUNetEdges;
    int2* dummyList;
    int2 *Male;
    int2 *Female;
    int2 *Coed;
    bool* isInfected;
   
    hipMallocManaged(&agentList,sizeof(agent)*population);
    hipMallocManaged (&Male , popWidth*sizeof(int2));
    hipMallocManaged (&Female , popWidth*sizeof(int2));
    hipMallocManaged (&Coed , population *sizeof(int2));
    hipMallocManaged (&dummyInts , population *sizeof(int));
    //initialize seed and allcoate space for random number values
    hiprandState * states;
    hipMallocManaged(&states,sizeof(hiprandState)* population );
    seed = generateSeed();
    setupKernel<<<1024,1024>>>(states, seed);
    hipDeviceSynchronize();

    //use sliding window to initialize all agent values
    hipMallocManaged(&hnrD, sizeof(heteroNetRanges));
    hipMallocManaged(&msmrD, sizeof(MSMNetRanges));
    hipMallocManaged(&idurD, sizeof(IDUNetRanges));
    hipMallocManaged(&users, sizeof(int)*population);
    hipMallocManaged(&isInfected,sizeof(bool)* population);


    hipMemcpy(hnrD, hnrH, sizeof(heteroNetRanges), hipMemcpyHostToDevice);
    hipMemcpy(msmrD, msmrH, sizeof(MSMNetRanges), hipMemcpyHostToDevice);
    hipMemcpy(idurD, idurH, sizeof(IDUNetRanges), hipMemcpyHostToDevice);
    
    thrust::fill(isInfected, isInfected+population, false);
    //initialize all agents

    agentsInit<<<windowSize,windowSize>>>(states, agentList, hnrD, msmrD, idurD, isInfected, users);
    hipDeviceSynchronize();
    numUsers = population - thrust::count(users, users+population, -1);
    
    printf("population %d \nnum users %d\n",population, numUsers);
    random<<<windowSize, windowSize>>>(Male, Female,Coed, time(0), 225);
    hipDeviceSynchronize();

    //if(thrust::count(isInfected, isInfected+population, true) < population * .0003){
        int max = population * .0001;
        for (int i = 0; i < 500; i++ ){
            int num = rand()% population;
            agentList[num].disease.isInfected = true;
        }
    //}

    thrust::fill(dummyInts, dummyInts+population, -1);
    thrust::copy_if(thrust::device, users, users+population, dummyInts, compressUser());
    numUsers = population - thrust::count(dummyInts, dummyInts + population, -1);
    thrust::fill(users, users + population, -1);
    thrust::copy_n(users,numUsers, dummyInts);
    printf("new num users %d\n", numUsers);

    hipFree(isInfected);
    hipFree(hnrD);
    hipFree(msmrD);
    hipFree(idurD);

    //create list of edges initilized to 0
    hipMallocManaged(&hNetEdges, sizeof(int2) * population);
    hipMallocManaged(&MSMNetEdges, sizeof(int2) * population);
    hipMallocManaged(&IDUNetEdges, sizeof(int2) * population);
    hipMallocManaged(&dummyList, sizeof(int2)*population);
    int2 init;
    init.x = -1;
    init.y = -1;
    thrust::fill(hNetEdges, hNetEdges+population, init);
    thrust::fill(MSMNetEdges, MSMNetEdges+population, init);
    thrust::fill(IDUNetEdges, IDUNetEdges+population, init);
    thrust::fill(dummyList, dummyList+population, init);

    //assign a random person to each agent  
    //and fill networks until maxLinks is reached 
    hipMallocManaged(&coordD,sizeof(int)* population); 
    printf("the number of Max links Hetero = %d\n", maxLinksHetero);
    printf("the number of Max links MSM = %d\n", maxLinksMSM);
    printf("the number of Max links IDU = %d\n", maxLinksIDU);
    while(timeStep  < 5000){
        //addlinks to hetero network
        while(numLinksHetero < maxLinksHetero){
            //choose random agent for every person hetero
            chooseRandAgent<<<windowSize,windowSize>>>(states, coordD, hetero, users, numUsers);
            hipDeviceSynchronize();
            fillHeteroNet<<<windowSize,windowSize>>>(states, hNetEdges, agentList, Male, Female, coordD);
            hipDeviceSynchronize();
            numLinksHetero = thrust::count_if(thrust::host,hNetEdges, hNetEdges+population, isEdge());
            thrust::copy_if(thrust::host, hNetEdges, hNetEdges + population, hNetEdges, compressNet());
        }

        //addlinks to MSM network
        while(numLinksMSM < maxLinksMSM){
            //select another agent from the list of males
            chooseRandAgent<<<windowSize,windowSize>>>(states, coordD, MSM,users, numUsers);
            hipDeviceSynchronize();
            fillMSMNet<<<windowSize,windowSize>>>(states, MSMNetEdges, agentList, Male, coordD);
            hipDeviceSynchronize();
            numLinksMSM = thrust::count_if(thrust::host,MSMNetEdges, MSMNetEdges+population, isEdge());
            thrust::copy_if(thrust::host, MSMNetEdges, MSMNetEdges + population, MSMNetEdges, compressNet());   
       
            }
        //addlinks to IDU network
        while(numLinksIDU < maxLinksIDU && numLinksIDU < numUsers){
            //select another agent from the list of males
            chooseRandAgent<<<windowSize,windowSize>>>(states, coordD, IDU,users, numUsers);
            hipDeviceSynchronize();
            fillIDUNet<<<windowSize,windowSize>>>(states, IDUNetEdges, agentList, coordD, users, numUsers);
            hipDeviceSynchronize();
            numLinksIDU = thrust::count_if(thrust::host,IDUNetEdges, IDUNetEdges+population, isEdge());
            thrust::copy_if(thrust::host, IDUNetEdges, IDUNetEdges + population, IDUNetEdges, compressNet());           
        }

        //if number of links formed is greater than maxLinks allowed remove some
        if(numLinksHetero > maxLinksHetero){
            trimNet<<<windowSize, windowSize>>>(states, hNetEdges, dummyList, maxLinksHetero, numLinksHetero);
            hipDeviceSynchronize();
            hipMemcpy(hNetEdges, dummyList, sizeof(int2)* population, hipMemcpyDeviceToDevice);
            thrust::fill(dummyList, dummyList+population, init);
        }
        if(numLinksMSM > maxLinksMSM){
            trimNet<<<windowSize, windowSize>>>(states, MSMNetEdges, dummyList, maxLinksMSM, numLinksMSM);
            hipDeviceSynchronize();
            hipMemcpy(MSMNetEdges, dummyList, sizeof(int2)* population, hipMemcpyDeviceToDevice);
            thrust::fill(dummyList, dummyList+population, init);
        }

        if(numLinksIDU > maxLinksIDU){
            trimNet<<<windowSize, windowSize>>>(states, IDUNetEdges, dummyList, maxLinksIDU, numLinksIDU);
            hipDeviceSynchronize();
            hipMemcpy(IDUNetEdges, dummyList, sizeof(int2)* population, hipMemcpyDeviceToDevice);
            thrust::fill(dummyList, dummyList+population, init);
            }

        //remove edges from networks if over a certain timestep
        if(timeStep % decayRate == 0){
            removeLinks<<<windowSize, windowSize>>>(states, hNetEdges, numLinksHetero, hetero);
            hipDeviceSynchronize();
            thrust::copy_if(thrust::host, hNetEdges, hNetEdges + population, hNetEdges, compressNet());
            removeLinks<<<windowSize, windowSize>>>(states, MSMNetEdges, numLinksMSM, MSM);
            hipDeviceSynchronize();
            removeLinks<<<windowSize, windowSize>>>(states, MSMNetEdges, numLinksIDU, IDU);
            hipDeviceSynchronize();
            numLinksHetero = thrust::count_if(thrust::host,hNetEdges, hNetEdges+population, isEdge());
            numLinksMSM = thrust::count_if(thrust::host,MSMNetEdges, MSMNetEdges+population, isEdge());
        }
        
        //update the infected population
        updateInfections<<<windowSize, windowSize>>>(states, agentList, hNetEdges, hetero);
        hipDeviceSynchronize();
        updateInfections<<<windowSize, windowSize>>>(states, agentList, MSMNetEdges, MSM);
        hipDeviceSynchronize();
        updateInfections<<<windowSize, windowSize>>>(states, agentList, IDUNetEdges, IDU);
        hipDeviceSynchronize();
        numInfected = thrust::count_if(thrust::host,agentList, agentList+population, infected());
        timeStep++;
        if(timeStep % 50 ==0){
            printf("the number of people infected after %d timestep is %d\n",timeStep, numInfected);
        }
        
    }    
       
    printf("gets here done\n");
    hipFree(hNetEdges);
    hipFree(coordD);
    hipFree(dummyList);
    hipFree(Male);
    hipFree(Female);
    hipFree(agentList);

    printf("%.f\n", seed);
    string word = "hello world";
    printf("%s\n", word.c_str());
    hipFree(states);
}

