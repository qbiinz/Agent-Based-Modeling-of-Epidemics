#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sstream>
#include <iostream>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include "AgentLib.h"
#include <thrust/fill.h>


using namespace std;

// generate seed for hiprand
extern double generateSeed();

//hiprand kernel setup
extern __global__ void setupKernel(hiprandState *state, double seed);

//uniform random kernel
extern __global__ void generateUniformKernel(hiprandState *state, double *result);

//choose agent iwth normal distributions
extern __global__ void chooseRandAgent(hiprandState *state,double *xCoord,double *yCoord, float std, int offset);

//select a random agent to calculate
extern void chooseRandAgent(hiprandState* states);

/*
TODO
uniformily distribute males and females in a list of len(population)
this way when agents are choosen at random then it is equally likely they can be added to either MSM or hetero networks
*/

int main(int argc, char* argv[]){
    double seed; 
    int n = 0;
    double *xCoord;
    double *yCoord;
    
    hipMallocManaged(&xCoord,sizeof(double)* windowSize* windowSize);
    hipMallocManaged(&yCoord,sizeof(double)* windowSize* windowSize);
    
    //initialize seed and allcoate space for random number values
    hiprandState * states;
    hipMallocManaged(&states,sizeof(hiprandState)* population );
    seed = generateSeed();
    setupKernel<<<1024,1024>>>(states, seed);
    hipDeviceSynchronize();

    //test 
    //generate random numbers
    // generateUniformKernel<<<64,64>>>(states,results);
    // hipDeviceSynchronize();
    
    //sliding window
    //while(links < maxLinks)
    while (windowSize * windowSize * n < population){
        chooseRandAgent<<<windowSize,windowSize>>>(states,xCoord, yCoord, 3.0, n);
        hipDeviceSynchronize();
        for (int i = 0; i < windowSize* windowSize; i++){
            printf("[%d, %d]\n",(int)xCoord[i], (int)yCoord[i]);
        }
        n++;
    } 
    /*
    
    TODO calculate probability that agentI and agentJ will be added to a network
    increment the number of links
    do this for all networks
    */

    printf("%.f\n", seed);
    string word = "hello world";
    printf("%s\n", word.c_str());


    hipFree(states);
    hipFree(xCoord);
    hipFree(yCoord);

}

